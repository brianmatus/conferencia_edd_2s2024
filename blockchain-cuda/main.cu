#include "elements/Blockchain.cuh"
#include <iostream>


int main() {
    Blockchain blockchain(6);
    blockchain.addBlock("hola");
    return 0;
}





#include <hip/hip_runtime.h>
#include <iostream>

// int main() {
//     hipDeviceProp_t prop;
//     hipGetDeviceProperties(&prop, 0); // Assuming device 0 (first GPU)
//
//     std::cout << "GPU: " << prop.name << std::endl;
//     std::cout << "Total Global Memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
//     std::cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << std::endl;
//     std::cout << "Max Threads Dim (Block Dimension): [" << prop.maxThreadsDim[0] << ", "
//               << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << "]" << std::endl;
//     std::cout << "Max Grid Size: [" << prop.maxGridSize[0] << ", "
//               << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << "]" << std::endl;
//     std::cout << "Warp Size: " << prop.warpSize << std::endl;
//     std::cout << "Number of SMs: " << prop.multiProcessorCount << std::endl;
//
//
//     // int blockSize;   // The launch configurator returned block size
//     // int minGridSize; // The minimum grid size needed to achieve the maximum occupancy
//     // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, myKernel, 0, 0);
//
//     return 0;
// }
