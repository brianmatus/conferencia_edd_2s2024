//
// Created by mrhax on 9/23/24.
//

#include "Blockchain.cuh"
#include "Block.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <cstdint>
#include "../utils/constants.hpp"
#include <chrono>

// Blockchain constructor
Blockchain::Blockchain(const int difficulty) : difficulty(difficulty) {
    const std::string s = "Genesis Block";
    char dataArr[MAX_DATA_SIZE] = {};
    memcpy(dataArr, s.c_str(), s.length());

    Block genesis_block(blockchain.size(), time(nullptr), dataArr);
    genesis_block.currentHash[0] = '0';
    blockchain.push_back(genesis_block);
}

// Add block to the blockchain
void Blockchain::addBlock( const std::string& data) {
    // Reset vars before launching the next kernel
    uint32_t h_stop_flag = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(stop_flag), &h_stop_flag, sizeof(uint32_t));

    uint32_t h_resulting_nonce = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(resulting_nonce), &h_resulting_nonce, sizeof(uint32_t));

    char dataArr[MAX_DATA_SIZE] = {};
    memcpy(dataArr, data.c_str(), data.length());

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///Block
    // Block h_block(blockchain.size(), time(nullptr), dataArr); //FIXME swap of real-time use
    Block h_block(blockchain.size(), 1727481184, dataArr);

    memcpy(h_block.previousBlockHash, blockchain.back().currentHash, 64);

    std::stringstream ss;
    ss << h_block.blockIndex  << "\n" << h_block.timeOfCreation<< "\n" << h_block.previousBlockHash<< "\n" << dataArr << "\n";
    const std::string resulting = ss.str();

    std::cout << "Resulting block: " << std::endl;
    std::cout << resulting << std::endl;

    char* d_block_data;
    hipMalloc(&d_block_data, MAX_DATA_SIZE);
    hipMemcpy(d_block_data, resulting.c_str(), resulting.length(), hipMemcpyHostToDevice);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///Output
    char h_output[65] = {};

    char* d_output;
    hipMalloc(&d_output, sizeof(char) * 65);
    hipMemset(d_output, 0, sizeof(char) * 65);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    const auto start = std::chrono::high_resolution_clock::now();
    hashKernel<<<MINING_SM_BLOCKS, MINING_BLOCK_THREADS>>>(d_block_data, MINING_TOTAL_THREADS, resulting.length(), d_output, difficulty);
    hipDeviceSynchronize();
    const auto end = std::chrono::high_resolution_clock::now();

    const std::chrono::duration<double> duration = end - start;
    std::cout << "Time taken: " << duration.count() << " seconds." << std::endl;

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipMemcpy(h_output, d_output, sizeof(char)*65, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&h_resulting_nonce, HIP_SYMBOL(resulting_nonce), sizeof(uint32_t));

    std::cout << "----------------------------------------------------" << std::endl;
    std::cout << "Resulting hash" << std::endl;
    std::cout << h_output << std::endl;
    std::cout << "For nonce:" << std::endl;
    std::cout << h_resulting_nonce << std::endl;

    hipFree(d_output);
    blockchain.push_back(h_block);
}
