#include "hip/hip_runtime.h"
//
// Created by mrhax on 9/23/24.
//

#include "Block.cuh"

#include <cstdio>

#include "../utils/sha256.cuh"
#include <cstring>

__device__ uint32_t stop_flag = 0;
__device__ uint32_t resulting_nonce = 0;

Block::Block(const uint32_t block_index, const time_t time_of_creation, const char* inputData) : blockIndex(block_index), timeOfCreation(time_of_creation) {
    verified_nonce = 0;
    valid_nonce = false;
    memset(previousBlockHash, 0, sizeof(previousBlockHash));
    memset(currentHash, 0, sizeof(currentHash));
    memset(data, 0, sizeof(inputData));
    memcpy(data, inputData, sizeof(inputData));

    // strcpy(previousBlockHash, "");
    // strcpy(data, "");
    // strcpy(currentHash, "");
}

__device__ void insert_nonce(char* device_input_data, uint32_t nonce, uint32_t nonce_insert_index) {
    // Break down the nonce into 4 bytes, little-endian
    device_input_data[nonce_insert_index]     = (char)(nonce & 0xFF);         // Least significant byte
    device_input_data[nonce_insert_index + 1] = (char)((nonce >> 8) & 0xFF);  // Next byte
    device_input_data[nonce_insert_index + 2] = (char)((nonce >> 16) & 0xFF); // Next byte
    device_input_data[nonce_insert_index + 3] = (char)((nonce >> 24) & 0xFF); // Most significant byte
}


__device__ bool check_leading_zeros(const char* hash, uint32_t num_zeros) {
    for (int i = 0; i < num_zeros; i++) {
        if (hash[i] != '0') {
            return false;
        }
    }
    return true;
}


__global__ void hashKernel(const char* device_input_data, const uint32_t nonce_increment, const uint32_t nonce_insert_index, char* output, const uint32_t difficulty) {
    const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t nonce = 0;
    uint32_t nonce_base = 0;

    char local_data[MAX_DATA_SIZE];
    for (int i = 0; i < MAX_DATA_SIZE; ++i) {
        local_data[i] = device_input_data[i];
    }

    while (atomicCAS(&stop_flag, 0, 0) == 0) {
        nonce = nonce_base + idx;
        insert_nonce(local_data, nonce, nonce_insert_index);

        char resulting_hash[65] = {};
        sha256(local_data, nonce_insert_index + 4, resulting_hash); // nonce_insert_index + 4 because of nonce of size uint32_t

        // printf("nonce:%i, resulting_hash:%s\n", nonce, resulting_hash);

        if (check_leading_zeros(resulting_hash, difficulty)) {
            if (atomicCAS(&stop_flag, 0, 1) == 0) {
                printf("----------------------------------------------\nBlock successfully mined by idx:%i nonce:%i\n%s\n", idx, nonce, local_data);
                for (int i = 0; i < 65; ++i) {
                    output[i] = resulting_hash[i];
                }
                atomicExch(&resulting_nonce, nonce);
            }
            break;
        }
        nonce_base += nonce_increment;
    }
}

__device__ uint32_t performHash(const uint32_t nonce, const char* data) {
    uint32_t hash = nonce;
    for (int i = 0; i < MAX_DATA_SIZE; ++i) {
        const char a = data[i];
        if (a == '\0') break;
        hash ^= a;
    }
    return hash;
}
